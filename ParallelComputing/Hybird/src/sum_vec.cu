#include "sum_vec.h"
#include "kernel.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

int get_max_block_threads()
{
    int dev_num;
    int max_threads;
    hipError_t status;

    status = hipGetDevice(&dev_num);

    status = hipDeviceGetAttribute(&max_threads, hipDeviceAttributeMaxThreadsPerBlock, dev_num);

    return max_threads;
}

void add(float* buffer_a, float* buffer_b, float* buffer_c, int n)
{
    hipError_t statusA; // records status of operations on A vec
    hipError_t statusB; // for B vec
    hipError_t statusC; // for C vec

    size_t size = n * sizeof(float); //size in bytes

    float *dev_a;
    float *dev_b;
    float *dev_c;
    statusA = hipMalloc(&dev_a, size);
    statusB = hipMalloc(&dev_b, size);
    statusC = hipMalloc(&dev_c, size);

    statusA = hipMemcpy(dev_a, buffer_a, size, hipMemcpyHostToDevice);
    statusB = hipMemcpy(dev_b, buffer_b, size, hipMemcpyHostToDevice);

    int block_size = get_max_block_threads();
    int blocks = n / block_size + (n % block_size > 0 ? 1 : 0);

    vec_add<<<blocks, block_size>>>(dev_a, dev_b, dev_c, n);
    
    statusC = hipMemcpy(buffer_c, dev_c, size, hipMemcpyDeviceToHost);
}
